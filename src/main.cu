#include "hip/hip_runtime.h"
#include <iostream>
#include "pch.cuh"

char *FILENAME;
int SCENE;
int width, height;
GPUShape **device_shapesPtrs = NULL;

extern __global__ void fillRedKernel(unsigned char *, int);

int main(int arc, char** argv) {

	if (arc != 4) {
		std::cerr << "Usage: ./A6 <SCENE> <IMAGE SIZE> <OUTFILENAME>";
		return 1;
	}

	// Init user input
	SCENE = atoi(argv[1]);
	width = atoi(argv[2]);
	height = width;
	FILENAME = argv[3];

	// Create Image
	Image image(width, height);

	// Allocate memory on the device
	unsigned char *d_pixels = NULL;
	size_t numPixels = width * height;
	size_t bufferSize = numPixels * sizeof(unsigned char) * 3;
	hipMalloc(&d_pixels, bufferSize);

	// Launch kernel
	int numThreads = 256;
	int blocks = (numPixels + numThreads - 1) / numThreads;
	fillRedKernel <<<blocks, numThreads >>> (d_pixels, numPixels);
	hipDeviceSynchronize();

	// 3) Copy back into the Image's vector
	hipMemcpy(image.getPixels(), d_pixels, bufferSize, hipMemcpyDeviceToHost);
	hipFree(d_pixels);

	// 4) Write out
	image.writeToFile("red_from_cuda.png");
	std::cout << "Wrote red_from_cuda.png\n";
	return 0;
}