#include "hip/hip_runtime.h"
#include "pch.cuh"
#include "Host.h"
#include "HostLauncher.h"

char *FILENAME;
int SCENE;
int width, height;
std::vector<GPUShape> hostShapes;
std::vector<GPULight> hostLights;
GPUShape *device_shapes;
GPUShape **device_shapesPtrs;
GPULight *device_lights;
GPUCamera *device_cam;
GPUMaterial *materials;
GPUMaterial *device_materials;

extern __global__ void fillRedKernel(unsigned char *, int);

int main(int arc, char** argv) {

	if (arc != 4) {
		printf("Usage: ./A6 <SCENE> <IMAGE SIZE> <OUTFILENAME>");
		return 1;
	}

	// Init user input
	SCENE = atoi(argv[1]);
	width = atoi(argv[2]);
	height = width;
	FILENAME = argv[3];
	mat4 IdMat = mat4();

	// Create Image
	Image image(width, height);

	// Create materials and allocate memory on device
	GPUMaterial hostMaterials[6];
	initMaterials(hostMaterials);
	device_materials = nullptr;
	hipMalloc(&device_materials, 6 * sizeof(GPUMaterial));
	hipMemcpy(device_materials, hostMaterials, 6 * sizeof(GPUMaterial), hipMemcpyHostToDevice);

	// Create shape device memory
	loadHostShapes(hostShapes, hostMaterials);
	device_shapes = nullptr;
	hipMalloc(&device_shapes, hostShapes.size() * sizeof(GPUShape));
	hipMemcpy(device_shapes, hostShapes.data(), hostShapes.size() * sizeof(GPUShape), hipMemcpyHostToDevice);

	// Array of pointers
	std::vector<GPUShape *> hostShapesPtrs(hostShapes.size());
	for (size_t i = 0; i < hostShapes.size(); i++) {
		hostShapesPtrs[i] = device_shapes + i;
	}
	// Copy to device
	device_shapesPtrs = nullptr;
	hipMalloc(&device_shapesPtrs, hostShapesPtrs.size() * sizeof(GPUShape *));
	hipMemcpy(device_shapesPtrs, hostShapesPtrs.data(), hostShapesPtrs.size() * sizeof(GPUShape *), hipMemcpyHostToDevice);

	// Create device friendly world light, one in this case and allocate memory on device
	GPULight worldLight = GPULight(vec3(-2.0f, 1.0f, 1.0f), vec3(1.0f), 1.0f);
	hostLights.push_back(worldLight);
	device_lights = nullptr;
	hipMalloc(&device_lights, hostLights.size() * sizeof(GPULight));
	hipMemcpy(device_lights, hostLights.data(), hostLights.size() * sizeof(GPULight), hipMemcpyHostToDevice);

	// Create device friendly camera, and to device friendly structs
	GPUCamera hostCamera;
	hostCamera.position = (vec3(0.0f, 0.0f, 5.0f));
	hostCamera.setTarget(vec3(0.0f)); // Look at origin
	hostCamera.setFOV(GPUradians(60.0f));
	hostCamera.worldUp = vec3(0.0f, 1.0f, 0.0f);
	device_cam = nullptr;
	hipMalloc(&device_cam, sizeof(GPUCamera));
	hipMemcpy(device_cam, &hostCamera, sizeof(GPUCamera), hipMemcpyHostToDevice);

	// Allocate memory on the device
	unsigned char *d_pixels = NULL;
	size_t numPixels = width * height;
	size_t bufferSize = numPixels * sizeof(unsigned char) * 3;
	hipMalloc(&d_pixels, bufferSize);
	// Launch kernel for scene , thanks GPT for these two vals
	int numThreads = 256;
	int blocks = (numPixels + numThreads - 1) / numThreads;
	//HAsceneOne(blocks, numThreads, d_pixels, numPixels, width, height, hostMaterials, device_shapesPtrs, device_lights, device_cam, FILENAME);
	KernelGenScenePixels <<<blocks, numThreads >>> (d_pixels, numPixels, width, height, device_cam, device_shapesPtrs, 3, device_lights, 1, 1, IdMat);
	hipDeviceSynchronize();

	// 3) Copy back into the Image's vector
	hipMemcpy(image.getPixels(), d_pixels, bufferSize, hipMemcpyDeviceToHost);
	hipFree(d_pixels);

	// 4) Write out
	image.writeToFile(FILENAME);
	printf("Wrote %s\n", FILENAME);
	return 0;
}